#include "hip/hip_runtime.h"
/*
.cu�ļ���CUDA c++�ļ�������һ����ֵ��﷨����kernel���������ǵ���gpu�̵߳ĺ�������

����dispatch GPU threads��host����ʵ�������cu�ļ������Ҫ��extern "C" ǰ׺

Ȼ����ISO c++����Ҫдһ��ͬ����extern "C"�������������ҿ�����.cpp����������

.cu�����host����������CUDA��gpu����ڵ�ɣ�

*/
#pragma warning (disable : 4819)

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

const int cBlockWidth = 16;
const int cBlockHeight = 16;
const int cBytePerPixel = 3;

__global__ void cudaKernel_RotateAndScale(const unsigned char* srcImage, unsigned char* destImage, int width, int height,int pitch,float angle,float scaleFactor)
{
	//blockIdx --- thread group index
	//thread index --- thread index with each thread group/block
	int pixelX = blockIdx.x * cBlockWidth + threadIdx.x;
	int pixelY = blockIdx.y * cBlockHeight + threadIdx.y;

	//********************************************************************
	float halfPixelWidth = float(width) / 2.0f;//��������Ĺ�һ��
	float halfPixelHeight = float(height) / 2.0f;
	//normalized X Y are mapped to [-1,1], centered at the center of screen
	float centeredPixelX = float(pixelX - halfPixelWidth) *(1.0f / scaleFactor);
	float centeredPixelY = float(halfPixelHeight - pixelY)*(1.0f / scaleFactor);
	/*
	[cos	sin]	[x]
	[-sin	cos]	[y]
	*/

	//����ͼƬ���˴�������ϵ��������Ч���ɡ����ȡ����Լ��ú�����
	float centeredRotatedPixelX = -(centeredPixelX  * cos(-angle) - centeredPixelY * sin(-angle));
	float centeredRotatedPixelY = -(centeredPixelX * sin(-angle) + centeredPixelY * cos(-angle));

	//fRotatedPixelX��Y�ĳ���scaleFactor������ķ���
	float fRotatedPixelX = (centeredRotatedPixelX + halfPixelWidth) / 2.0f / scaleFactor;
	float fRotatedPixelY = (halfPixelHeight - centeredRotatedPixelY) / 2.0f / scaleFactor;

	//�����ǰ������ת��û�г��磨��ζ�ſ��Բ�����
	unsigned char sampleColor[3];
	if (fRotatedPixelX >= 0 && fRotatedPixelX < width - 1 &&
		fRotatedPixelY >= 0 && fRotatedPixelY < height - 1)
	{
		int rotatedPixelX = int(fRotatedPixelX);
		int rotatedPixelY = int(fRotatedPixelY);
		unsigned char c1[3], c2[3], c3[3], c4[3];//4��RGB color

		for (int i = 0; i < 3; ++i)c1[i] = srcImage[(rotatedPixelY +0) * pitch + (rotatedPixelX +0)* cBytePerPixel + 2 - i];
		for (int i = 0; i < 3; ++i)c2[i] = srcImage[(rotatedPixelY +0) * pitch + (rotatedPixelX +1) * cBytePerPixel + 2 - i];
		for (int i = 0; i < 3; ++i)c3[i] = srcImage[(rotatedPixelY +1) * pitch + (rotatedPixelX +0) * cBytePerPixel + 2 - i];
		for (int i = 0; i < 3; ++i)c4[i] = srcImage[(rotatedPixelY +1) * pitch + (rotatedPixelX +1) * cBytePerPixel + 2 - i];

		//��ֵϵ��
		float t1 = fRotatedPixelX - float(rotatedPixelX);
		float t2 = fRotatedPixelY - float(rotatedPixelY);

		//Hermite���ײ�ֵ
		auto Hermite = [](float t, const unsigned char c1[3], const unsigned char c2[3],unsigned char c3[3] )
		{
			//hermite��ֵ�ľ����  : 2|x|^3 - 3|x|^2 +1���ѿ������ھ���˺�����ԭ��
			float factor1 = 2.0f * t * t * t - 3.0f * t * t + 1.0f;
			float factor2 = 2.0f * (1.0f - t) * (1.0f - t) * (1.0f - t) - 3.0f * (1.0f - t) * (1.0f - t) + 1.0f;
			c3[0] = c1[0] * factor1 + c2[0]*factor2;
			c3[1] = c1[1] * factor1 + c2[1]*factor2;
			c3[2] = c1[2] * factor1 + c2[2]*factor2;
		};

		unsigned char tmp1[3], tmp2[3];
		Hermite(t1, c1, c2, tmp1);
		Hermite(t1, c3, c4, tmp2);

		Hermite(t2, tmp1, tmp2, sampleColor);
	}
	else
	{
		//����ĸ��ұ�ںð�
		sampleColor[0] = 0;
		sampleColor[1] = 0;
		sampleColor[2] = 0;
	}


	//********************************************************************
	//���
	destImage[pixelY * pitch + pixelX *cBytePerPixel + 2] = sampleColor[0];//r
	destImage[pixelY * pitch + pixelX *cBytePerPixel + 1] = sampleColor[1];//g
	destImage[pixelY * pitch + pixelX *cBytePerPixel + 0] = sampleColor[2];//b
}

/*entry point of CUDA*/
extern "C" int cudaHost_RotateAndScale(const unsigned char* srcImage, unsigned char* destImage, int width,int height, int pitch, float angle, float scaleFactor)
{
	hipError_t err = hipSuccess;

	//������֪�������ʲô�ã�
	hipSetDevice(0);

	//�����Դ棬�Դ��ߴ�Ҫ���ڴ��е�һ��
	unsigned char* pDeviceSrcData =nullptr;
	unsigned char* pDeviceDestData = nullptr;

	//MFC��CImage��pitch��Ȼ�Ǹ��ģ�����������������������
	//���ٷ�����һ���𣿣���
	int imageByteSize = height * pitch;

	hipMalloc((void**)&pDeviceSrcData, imageByteSize);
	err = hipGetLastError();
	if (err != hipSuccess)return 11;

	hipMalloc((void**)&pDeviceDestData, imageByteSize);
	err = hipGetLastError();
	if (err != hipSuccess)return 12;

	//�ڴ������update���Դ�
	hipMemcpy(pDeviceSrcData, srcImage, imageByteSize , hipMemcpyHostToDevice);
	err = hipGetLastError();
	if (err != hipSuccess)return 13;

	//���ģ���ú˺���(ÿ��gpu thread��Ҫִ��kernel function)
	//������kernel function���棬src dataִ�����߼���д��Dest data ����
	//���������� �� <<<blocksPerGrid, threadsPerBlock>>>

	dim3 dimGrid(width / cBlockWidth, height / cBlockHeight);	//����(grid)��ά�ȣ�grid�ĵ�Ԫ���߳���
	dim3 dimBlock(cBlockWidth, cBlockHeight);	//�߳���(Thread Group)/��(block)�ĳߴ�
	cudaKernel_RotateAndScale << <dimGrid, dimBlock >> >
		(pDeviceSrcData, pDeviceDestData,width,height,pitch,angle,scaleFactor);
	err = hipGetLastError();
	if (err != hipSuccess)return 14;

	//gpu threads��ͬ���ȴ�
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess)return 15;

	//��ȡ�����copy���ڴ�
	hipMemcpy(destImage, pDeviceDestData, imageByteSize,  hipMemcpyDeviceToHost);
	err = hipGetLastError();
	if (err != hipSuccess)return 16;

	//??��֪�������reset device??
	hipDeviceReset();
	err = hipGetLastError();
	if (err != hipSuccess)return 16;

	return 0;
}