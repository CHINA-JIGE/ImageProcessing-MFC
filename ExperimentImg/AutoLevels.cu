#include "hip/hip_runtime.h"
/*
.cu�ļ���CUDA c++�ļ�������һ����ֵ��﷨����kernel���������ǵ���gpu�̵߳ĺ�������

����dispatch GPU threads��host����ʵ�������cu�ļ������Ҫ��extern "C" ǰ׺

Ȼ����ISO c++����Ҫдһ��ͬ����extern "C"�������������ҿ�����.cpp����������

.cu�����host����������CUDA��gpu����ڵ�ɣ�

*/
#pragma warning (disable : 4819)

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

const int cBlockWidth = 16;
const int cBlockHeight = 16;
const int cBytePerPixel = 3;

typedef unsigned char byte;

__global__ void cudaKernel_AutoLevels(
	const unsigned char* srcImage, unsigned char* destImage, int pitch,
	byte minR, byte maxR,
	byte minG, byte maxG,
	byte minB, byte maxB
	)
{
	//blockIdx --- thread group index
	//thread index --- thread index with each thread group/block
	int pixelX = blockIdx.x * cBlockWidth + threadIdx.x;
	int pixelY = blockIdx.y * cBlockHeight + threadIdx.y;

	//********************************************************************
	byte c[3];
	c[0] = srcImage[pixelY * pitch + pixelX* cBytePerPixel + 2];
	c[1] = srcImage[pixelY * pitch + pixelX* cBytePerPixel + 1];
	c[2] = srcImage[pixelY * pitch + pixelX* cBytePerPixel + 0];

	byte rangeR = maxR - minR;
	byte rangeG = maxG - minG;
	byte rangeB = maxB - minB;

	if (rangeR != 0)
	{
		float ratioR = float(c[0] - minR) / rangeR;
		if (ratioR >= 0.0f && ratioR <= 1.0f) c[0] = byte(255.0f * ratioR);
	}

	if (rangeG != 0)
	{
		float ratioG = float(c[1] - minG) / rangeG;
		if (ratioG >= 0.0f && ratioG <= 1.0f)c[1] = byte(255.0f * ratioG);
	}

	if (rangeB != 0)
	{
		float ratioB = float(c[2] - minB) / rangeB;
		if (ratioB >= 0.0f && ratioB <= 1.0f)c[2] = byte(255.0f * ratioB);
	}

	//********************************************************************
	//���
	destImage[pixelY * pitch + pixelX *cBytePerPixel + 2] = c[0];//r
	destImage[pixelY * pitch + pixelX *cBytePerPixel + 1] = c[1];//g
	destImage[pixelY * pitch + pixelX *cBytePerPixel + 0] = c[2];//b
}

/*entry point of CUDA*/
extern "C" int cudaHost_AutoLevels(const unsigned char* srcImage, unsigned char* destImage, 
	int width, int height, int pitch,
	byte minR, byte maxR,
	byte minG, byte maxG,
	byte minB, byte maxB
	)
{
	hipError_t err = hipSuccess;

	//������֪�������ʲô�ã�
	hipSetDevice(0);

	//�����Դ棬�Դ��ߴ�Ҫ���ڴ��е�һ��
	unsigned char* pDeviceSrcData = nullptr;
	unsigned char* pDeviceDestData = nullptr;

	//MFC��CImage��pitch��Ȼ�Ǹ��ģ�����������������������
	//���ٷ�����һ���𣿣���
	int imageByteSize = height * pitch;

	hipMalloc((void**)&pDeviceSrcData, imageByteSize);
	err = hipGetLastError();
	if (err != hipSuccess)return 11;

	hipMalloc((void**)&pDeviceDestData, imageByteSize);
	err = hipGetLastError();
	if (err != hipSuccess)return 12;

	//�ڴ������update���Դ�
	hipMemcpy(pDeviceSrcData, srcImage, imageByteSize, hipMemcpyHostToDevice);
	err = hipGetLastError();
	if (err != hipSuccess)return 13;

	//���ģ���ú˺���(ÿ��gpu thread��Ҫִ��kernel function)
	//������kernel function���棬src dataִ�����߼���д��Dest data ����
	//���������� �� <<<blocksPerGrid, threadsPerBlock>>>

	dim3 dimGrid(width / cBlockWidth, height / cBlockHeight);	//����(grid)��ά�ȣ�grid�ĵ�Ԫ���߳���
	dim3 dimBlock(cBlockWidth, cBlockHeight);	//�߳���(Thread Group)/��(block)�ĳߴ�
	cudaKernel_AutoLevels << <dimGrid, dimBlock >> >
		(pDeviceSrcData, pDeviceDestData, pitch, minR, maxR, minG, maxG, minB, maxB);
	err = hipGetLastError();
	if (err != hipSuccess)return 14;

	//gpu threads��ͬ���ȴ�
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess)return 15;

	//��ȡ�����copy���ڴ�
	hipMemcpy(destImage, pDeviceDestData, imageByteSize, hipMemcpyDeviceToHost);
	err = hipGetLastError();
	if (err != hipSuccess)return 16;

	//??��֪�������reset device??
	hipDeviceReset();
	err = hipGetLastError();
	if (err != hipSuccess)return 16;

	return 0;
}